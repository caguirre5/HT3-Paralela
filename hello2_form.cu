/*
 ============================================================================
 Author        : C. Aguirre
 Version       : 1.0
 Last modified : November 2023
 License       : Released under the GNU GPL 3.0
 Description   : CUDA program that prints "Hello world" from maximun global ID along with name and ID
 To build use  : nvcc <program_name>.cu -o <name> -arch=sm_75
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello()
{
    int myID = (blockIdx.z * gridDim.x * gridDim.y +
                blockIdx.y * gridDim.x +
                blockIdx.x) *
                   blockDim.x * blockDim.y * blockDim.z +
               threadIdx.z * blockDim.x * blockDim.y +
               threadIdx.y * blockDim.x +
               threadIdx.x;

    // // Calculate the global thread ID
    // int myID = ( blockIdx.z * gridDim.x * gridDim.y +
    //              blockIdx.y * gridDim.x +
    //              blockIdx.x ) * blockDim.x +
    //              threadIdx.x;

    printf("Hello world from thread %i: - Cristian Aguirre 20231\n", myID);
}

int main()
{
    dim3 g(4, 2);
    dim3 b(32, 16);
    hello<<<g, b>>>();
    hipDeviceSynchronize();
    return 0;
}
