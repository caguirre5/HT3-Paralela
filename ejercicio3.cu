/*
 ============================================================================
 Author        : C. Aguirre
 Version       : 1.0
 Last modified : November 2023
 License       : Released under the GNU GPL 3.0
 Description   : CUDA program that prints "Hello world" from maximun global ID along with name and ID
 To build use  : nvcc <program_name>.cu -o <name> -arch=sm_75
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void process_data()
{
    int globalID = blockIdx.x * blockDim.x + threadIdx.x;
    int maxGlobalID = gridDim.x * blockDim.x - 1; // ID global máximo

    if (globalID == maxGlobalID)
    {
        printf("Hello world the maximum global ID %i: - Cristian Aguirre 20231\n", globalID);
    }
}

int main()
{
    int numData = 100000;
    int blockSize = 1024;

    int numBlocks = (numData + blockSize - 1) / blockSize;

    dim3 blocks(numBlocks, 1);
    dim3 threadsPerBlock(blockSize, 1);

    process_data<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();

    return 0;
}
