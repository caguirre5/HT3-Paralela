#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : C. Aguirre
 Version       : 1.0
 Last modified : November 2023
 License       : Released under the GNU GPL 3.0
 Description   : CUDA program that prints "Hello world" from maximun global ID along with name and ID
 To build use  : nvcc <program_name>.cu -o <name> -arch=sm_75
 ============================================================================
 */

#include <stdio.h>

__global__ void hello()
{
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello world from thread %d in block %d\n", threadID, blockIdx.x);
    if (threadID == 1023)
    {
        printf("ThreadID 1023 - Nombre: Cristian Aguirre, Carnet: 20231\n");
    }
}

int main()
{
    hello<<<2, 1024>>>();
    hipDeviceSynchronize(); // deprecated
    return 0;
}